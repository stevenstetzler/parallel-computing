#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

const unsigned int N_TRIALS = 10;

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) { 
    //printf("%s\n", cudaGetErrorString(e));
    if (e != hipSuccess) {
        printf("\nCUDA error: %s\n", hipGetErrorString(e));
        exit(1);
    }
}

__global__ void matrix_mult_gpu(float* A, float* B, float* C, int N)
{
    //printf("Hi Cuda World\n");
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("row = %d, col = %d\n", row, col);
    // printf("A[0] = %d, B[0] = %d\n", A[0], B[0]);
    
    int k, idx;
    float sum;   
 
    if (row < N && col < N) {
        idx = row * N + col;
        sum = 0;
        for (k = 0; k < N; k++) {
            //if (row == 99 && col == 39) {
            //    printf("k = %d\tA = %f\tB = %f\n", k, A[row * N + k], B[k * N + col]);
            //}
            sum += A[row * N + k] * B[k * N + col];
        }
        //printf("row = %d, col = %d, sum = %f\n", row, col, sum);
        C[idx] = sum;
        //if (row == 99 && col == 39) {
        //    printf("sum = %f\n", sum);
        //}
    } 
}

void matrix_mult_cpu(float* A, float* B, float* C, int N) 
{
    int row, col, k, idx;
    float sum;

    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            idx = row * N + col;
            sum = 0;
            for (k = 0; k < N; k++) {
                //if (row == 99 && col == 39) {
                //    printf("k = %d\tA = %f\tB = %f\n", k, A[row * N + k], B[k * N + col]);
                //}
                sum += A[row * N + k] * B[k * N + col];
            }
            //printf("row = %d, col = %d, sum = %f\n", row, col, sum);
            C[idx] = sum;
            //if (row == 99 && col == 39) {
            //    printf("sum = %f\n", sum);
            //}
        }
    }
}

void compareHostAndGpuOutput(float* C, float* C_CPU, int N) {
    int totalElements = N * N;
    int missmatchCount = 0;
    for (int i = 0; i < totalElements; i++) {
        // printf("%d\n",i);
        if (fabs(C[i] - C_CPU[i]) > 0.01) {
            missmatchCount++;
            printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
        }
    }
    if (missmatchCount > 0) {
        printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
    } else {
        printf("Computation is correct: CPU and GPU outputs match\n");
    }
}


int main( int argc, char** argv )
{
    int N = (argc > 1) ? atoi(argv[1]) : 100;
    int check_cpu = (argc > 2) ? atoi(argv[2]) : 0;

    printf("N = %d\nGrid: %d x %d\nThreads: %d x %d\n", N, (int) ceil((double) N / 32), (int) ceil((double) N / 32), 32, 32);

    dim3 block_per_grid((int) ceil((double) N / 32), (int) ceil((double) N / 32) , 1);
    dim3 thread_per_block(32, 32, 1);
    
    float* A = (float*) malloc(N * N * sizeof(float));
    float* B = (float*) malloc(N * N * sizeof(float));
    float* C = (float*) malloc(N * N * sizeof(float));

    int i, j;
    float val;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            val = (rand() % 1000) * 0.001;
            A[i * N + j] = val;
            B[i * N + j] = val;
            C[i * N + j] = val;
            //printf("%f\t", A[i * N + j]);
        }
        //printf("\n");
    }

    float* A_GPU; // = (int*) malloc(N * N * sizeof(int));
    float* B_GPU; // = (int*) malloc(N * N * sizeof(int));
    float* C_GPU;

    clock_t start, end;
    double elapsed;

    if (check_cpu) {
        start = clock();
        matrix_mult_cpu(A, B, C, N);
        end = clock();
        
        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        printf("CPU: %.10f seconds\n", elapsed);
    }

    double copy_avg_time = 0;
    double comp_avg_time = 0;

    for (i = 0 ; i < N_TRIALS; i++) {
        start = clock();

        check_error(hipMalloc((void **) &A_GPU, N * N * sizeof(float)));
        check_error(hipMalloc((void **) &B_GPU, N * N * sizeof(float)));
        check_error(hipMalloc((void **) &C_GPU, N * N * sizeof(float)));
        check_error(hipMemcpy(A_GPU, A, N * N * sizeof(float), hipMemcpyHostToDevice));
        check_error(hipMemcpy(B_GPU, B, N * N * sizeof(float), hipMemcpyHostToDevice));
        check_error(hipMemcpy(C_GPU, C, N * N * sizeof(float), hipMemcpyHostToDevice));

        end = clock();

        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        copy_avg_time += elapsed; 
        //printf("GPU_copy: %.10f seconds\n", elapsed);
        
        start = clock();
        
        matrix_mult_gpu<<<block_per_grid, thread_per_block>>>(A_GPU, B_GPU, C_GPU, N);
        hipDeviceSynchronize();
        
        end = clock();

        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        comp_avg_time += elapsed; 
        
        //printf("GPU: %.10f seconds\n", elapsed);

        if (i != N_TRIALS - 1) {
            check_error(hipFree(A_GPU));
            check_error(hipFree(B_GPU));
            check_error(hipFree(C_GPU));
        }
    }
    
    printf("GPU_copy: %.10f seconds\n", copy_avg_time / N_TRIALS);
    printf("GPU: %.10f seconds\n", comp_avg_time / N_TRIALS);

    if (check_cpu) {
        float* C_GPU_Copy = (float*) malloc(N * N * sizeof(float));

        check_error(hipMemcpy(C_GPU_Copy, C_GPU, N * N * sizeof(float), hipMemcpyDeviceToHost));
        compareHostAndGpuOutput(C, C_GPU_Copy, N);
    }

    return 0;
}

