/*
 * CS 4444
 * Steven Stetzler
 * Homework 5: Matrix-Matrix Multiplication with CUDA
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) { 
    if (e != hipSuccess) {
        printf("\nCUDA error: %s\n", hipGetErrorString(e));
        exit(1);
    }
}

// A GPU implementation of matrix multiplication.
// Given three N x N matrices A, B, and C we compute C = A x B
__global__ void matrix_mult_gpu(float* A, float* B, float* C, int N) {
    // Get the row and column of C that this thread should work on
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int k, idx;
    float sum;   
    
    // Ignore threads that would compute values outside of the boundary of the matrix
    if (row < N && col < N) {
        idx = row * N + col;
        sum = 0;
        for (k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[idx] = sum;
    } 
}

// A naive (no cache blocking) CPU implementation of matrix multiplication
// Given three N x N matrices A, B, and C we compute C = A x B
void matrix_mult_cpu(float* A, float* B, float* C, int N) {
    int row, col, k, idx;
    float sum;

    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            idx = row * N + col;
            sum = 0;
            for (k = 0; k < N; k++) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[idx] = sum;
        }
    }
}

// Compare the values in two N x N matrices C and C_CPU
void compareHostAndGpuOutput(float* C, float* C_CPU, int N) {
    int totalElements = N * N;
    int missmatchCount = 0;
    for (int i = 0; i < totalElements; i++) {
        if (fabs(C[i] - C_CPU[i]) > 0.01) {
            missmatchCount++;
            printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
        }
    }
    if (missmatchCount > 0) {
        printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
    } else {
        printf("Computation is correct: CPU and GPU outputs match\n");
    }
}

// Main method
int main(int argc, char** argv) {
    // The problem size N is the dimension of the arrays
    int N = (argc > 1) ? atoi(argv[1]) : 100;
    // Option whether or not to check GPU output against CPU implementation
    // Should not be included for large N, as CPU will be very slow
    int check_cpu = (argc > 2) ? atoi(argv[2]) : 0;
    // Options to specify the thread gridding
    int thread_x = (argc > 3) ? atoi(argv[3]) : 32;
    int thread_y = (argc > 4) ? atoi(argv[4]) : 32;
    // Option to specify the number of trials for run time tests of the GPU
    int n_trials = (argc > 5) ? atoi(argv[5]) : 10;

    // Compute block gridding from the thread gridding
    // This is the minimum size block gridding given the size of the array and the thread gridding that guarantees that all values
    // in C will be computed
    int grid_x = (int) ceil((double) N / thread_x);
    int grid_y = (int) ceil((double) N / thread_y);

    // Print run parameters
    printf("N = %d\nGrid: %d x %d\nThreads: %d x %d\nTrials: %d\n", N, grid_x, grid_y, thread_x, thread_y, n_trials);

    // Specify block and thread gridding
    dim3 block_per_grid(grid_x, grid_y , 1);
    dim3 thread_per_block(thread_x, thread_y, 1);
    
    // Create and allocate three arrays
    float* A = (float*) malloc(N * N * sizeof(float));
    float* B = (float*) malloc(N * N * sizeof(float));
    float* C = (float*) malloc(N * N * sizeof(float));
    
    // Create pointers for GPU arrays, but do not allocate yet!
    float* A_GPU;
    float* B_GPU;
    float* C_GPU;

    // Perform random initialization of the arrays
    int i, j;
    float val;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            val = (rand() % 1000) * 0.001;
            A[i * N + j] = val;
            B[i * N + j] = val;
            C[i * N + j] = val;
        }
    }

    clock_t start, end;
    double elapsed;

    // If we want to check against CPU, perform CPU matrix multiplication and time it
    if (check_cpu) {
        start = clock();
        matrix_mult_cpu(A, B, C, N);
        end = clock();
        
        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        printf("CPU: %.10f seconds\n", elapsed);
    }

    double copy_avg_time = 0;
    double comp_avg_time = 0;

    // For each trial in run time analysis
    for (i = 0 ; i < n_trials; i++) {
        // Time the copy operation
        start = clock();

        // Allocate arrays on GPU
        check_error(hipMalloc((void **) &A_GPU, N * N * sizeof(float)));
        check_error(hipMalloc((void **) &B_GPU, N * N * sizeof(float)));
        check_error(hipMalloc((void **) &C_GPU, N * N * sizeof(float)));
        // Copy in values from A, B, and C
        check_error(hipMemcpy(A_GPU, A, N * N * sizeof(float), hipMemcpyHostToDevice));
        check_error(hipMemcpy(B_GPU, B, N * N * sizeof(float), hipMemcpyHostToDevice));
        check_error(hipMemcpy(C_GPU, C, N * N * sizeof(float), hipMemcpyHostToDevice));

        end = clock();

        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        copy_avg_time += elapsed; 
       
        // Time the computation operation 
        start = clock();
        
        // Perform GPU matrix multiply
        matrix_mult_gpu<<<block_per_grid, thread_per_block>>>(A_GPU, B_GPU, C_GPU, N);
        hipDeviceSynchronize();
        
        end = clock();

        elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        comp_avg_time += elapsed; 
        
        // Free arrays if this isn't our last trial
        if (i != n_trials - 1) {
            check_error(hipFree(A_GPU));
            check_error(hipFree(B_GPU));
            check_error(hipFree(C_GPU));
        }
    }

    // Print timing results    
    printf("GPU_copy: %.10f seconds\n", copy_avg_time / n_trials);
    printf("GPU: %.10f seconds\n", comp_avg_time / n_trials);

    // If we wanted to check against CPU, do so
    if (check_cpu) {
        // Copy result from GPU
        float* C_GPU_Copy = (float*) malloc(N * N * sizeof(float));
        check_error(hipMemcpy(C_GPU_Copy, C_GPU, N * N * sizeof(float), hipMemcpyDeviceToHost));
        // Compare GPU and CPU output
        compareHostAndGpuOutput(C, C_GPU_Copy, N);
    }

    return 0;
}

